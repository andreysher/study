/********************************************************************
 * BenchIT - Performance Measurement for Scientific Applications
 * Contact: developer@benchit.org
 *
 * For license details see COPYING in the package base directory
 *******************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "interface.h"
/*  Header for local functions
 */
#include "work.h"

/** These variables will help us to keep the overview over the arrays
  * we access for our functions/data.
  */
/* Number of different ways an algorithm will be measured.
   Example: loop orders: ijk, ikj, jki, jik, kij, kji -> functionCount=6 with
   each different loop order in an own function. */
int functionCount = 3; //Latency
/* Number of fixed functions we have per measurement.
   Example: execution time and MFLOPS are measured for each loop order
   -> valuesPerFunction=2 */
int valuesPerFunction = 1;
int arrayLength;
int arraySize;
int iterations;

/*  Header for local functions
 */
void evaluate_environment(void);

/**  The implementation of the bi_getinfo from the BenchIT interface.
 *   Here the infostruct is filled with information about the
 *   kernel.
 *   @param infostruct  a pointer to a structure filled with zeros
 */

void bi_getinfo(bi_info * infostruct)
{
	 int index1;
   /* get environment variables for the kernel */
   evaluate_environment();
   infostruct->codesequence = bi_strdup("accessMem(cycles, count)");
   infostruct->xaxistext = bi_strdup("Accessed values");
   infostruct->num_measurements = infostruct->listsize;
   infostruct->num_processes = 1;
   infostruct->num_threads_per_process = 0;

   infostruct->kernel_execs_mpi1 = 0;
   infostruct->kernel_execs_mpi2 = 0;
   infostruct->kernel_execs_pvm = 0;
   infostruct->kernel_execs_omp = 0;
   infostruct->kernel_execs_pthreads = 0;

   infostruct->numfunctions = functionCount * valuesPerFunction;

   /* allocating memory for y axis texts and properties */
   allocYAxis(infostruct);
   /* setting up y axis texts and properties */
   for(index1 = 0; index1 < infostruct->numfunctions; index1++){
     infostruct->yaxistexts[index1] = bi_strdup("Cycles");
     infostruct->selected_result[index1] = SELECT_RESULT_LOWEST;
     infostruct->base_yaxis[index1] = 0;
     infostruct->legendtexts[index1] = bi_strdup("Latency");
   }
   infostruct->legendtexts[0] = bi_strdup("Same adress");
   infostruct->legendtexts[1] = bi_strdup("Same bank");
   infostruct->legendtexts[2] = bi_strdup("Succ. banks");
}

/** Implementation of the bi_init of the BenchIT interface.
 *  Here you have the chance to allocate the memory you need.
 *  It is also possible to allocate the memory at the beginning
 *  of every single measurement and to free the memory thereafter.
 *  But always making use of the same memory is faster.
 *  HAVE A LOOK INTO THE HOWTO !
 */
void* bi_init(int problemSizemax)
{
   mydata_t* mdp;
   mdp = (mydata_t*)malloc(sizeof(mydata_t));
   if (mdp == 0)
   {
      fprintf(stderr, "Allocation of structure mydata_t failed\n"); fflush(stderr);
      exit(127);
   }
   
   hipDeviceSetSharedMemConfig(hipSharedMemBankSizeFourByte);
   hipDeviceSetCacheConfig(hipFuncCachePreferShared);
   
   mdp->h_array = (uint*) malloc(arraySize);
   //Add 1 dummy value
   CUDA_CHECK(hipMalloc((void**)&(mdp->d_array),arraySize+sizeof(uint)));
   mdp->h_duration = (uint*) malloc(2*sizeof(uint));
   CUDA_CHECK(hipMalloc((void**)&(mdp->d_duration),2*sizeof(uint)));

   return (void*)mdp;
}

/** The central function within each kernel. This function
 *  is called for each measurement step seperately.
 *  @param  mdpv         a pointer to the structure created in bi_init,
 *                       it is the pointer the bi_init returns
 *  @param  problemSize  the actual problemSize
 *  @param  results      a pointer to a field of doubles, the
 *                       size of the field depends on the number
 *                       of functions, there are #functions+1
 *                       doubles
 *  @return 0 if the measurement was sucessfull, something
 *          else in the case of an error
 */
int bi_entry(void* mdpv, int problemSize, double* results)
{
	int i, j;
  /* cast void* pointer */
  mydata_t* mdp = (mydata_t*)mdpv;

  /* calculate real problemSize */
  problemSize = bi_get_list_element(problemSize); 
  
  // the xaxis value needs to be stored only once!
  results[0] = (double)problemSize;
  
	for(i=0; i < functionCount;i++){
  	for(j = 0; j < arrayLength; j++)
  		mdp->h_array[j] = 0;
		int stride;
		if(i == 0){
  		stride=0;
  	}else if(i==1){
  		stride=1;
  	}else{
  		stride=2;
  	}
  	
  	CUDA_CHECK(hipMemcpy(mdp->d_array,mdp->h_array,arraySize,hipMemcpyHostToDevice));
  	
  	dim3 dimBlock(1);
  	dim3 dimGrid(1);
  	CUDA_CHECK_KERNEL_SYNC(testLatencyN<<<dimGrid, dimBlock, arraySize>>>(mdp->d_array, arrayLength, problemSize, stride, iterations, mdp->d_duration));
  	CUDA_CHECK(hipMemcpy(mdp->h_duration,mdp->d_duration,2*sizeof(uint),hipMemcpyDeviceToHost));
  	results[i+1]=mdp->h_duration[0] / (1 * iterations);
	}
  return 0;
}

// Clean up the memory
void bi_cleanup(void* mdpv)
{
   mydata_t* mdp = (mydata_t*)mdpv;
   free(mdp->h_array);
   free(mdp->h_duration);
   CUDA_CHECK(hipFree(mdp->d_array));
   CUDA_CHECK(hipFree(mdp->d_duration));
   free(mdp);
   return;
}
/********************************************************************/
/*************** End of interface implementations *******************/
/********************************************************************/

/* Reads the environment variables used by this kernel. */
void evaluate_environment()
{
   int errors = 0;
   char * p = 0;

   p = bi_getenv("BENCHIT_KERNEL_ACCESSCOUNT", 0);
   if(p==0) errors++;
   else bi_parselist(p);

   p = bi_getenv("BENCHIT_KERNEL_ARRAYLENGTH", 0);
   if(p==0) errors++;
   else arrayLength = atoi(p);
   arraySize = arrayLength * sizeof(uint);

   p = bi_getenv("BENCHIT_KERNEL_ITERATIONS", 0);
   if(p==0) errors++;
   else iterations = atoi(p);

   if (errors > 0)
   {
      fprintf(stderr, "There's at least one environment variable not set!\n");
      fprintf(stderr, "This kernel needs the following environment variables:\n");
      fprintf(stderr, "BENCHIT_KERNEL_ACCESSCOUNT\n");
      fprintf(stderr, "BENCHIT_KERNEL_ARRAYSIZE\n");
      fprintf(stderr, "BENCHIT_KERNEL_ITERATIONS\n");
      exit(1);
   }
}
