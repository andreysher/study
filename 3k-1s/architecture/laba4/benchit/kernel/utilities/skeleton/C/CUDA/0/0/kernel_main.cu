#include "hip/hip_runtime.h"
/********************************************************************
 * BenchIT - Performance Measurement for Scientific Applications
 * Contact: developer@benchit.org
 *
 * For license details see COPYING in the package base directory
 *******************************************************************/
/* Kernel: cuda kernel skeleton
 *******************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "interface.h"
/*  Header for local functions
 */
#include "work.h"

/** These variables will help us to keep the overview over the arrays
  * we access for our functions/data.
  */
/* Number of different ways an algorithm will be measured.
   Example: loop orders: ijk, ikj, jki, jik, kij, kji -> functionCount=6 with
   each different loop order in an own function. */
int functionCount;
/* Number of fixed functions we have per measurement.
   Example: execution time and MFLOPS are measured for each loop order
   -> valuesPerFunction=2 */
int valuesPerFunction;

int MIN, MAX, INCREMENT;

/*  Header for local functions
 */
void evaluate_environment(void);

/**  The implementation of the bi_getinfo from the BenchIT interface.
 *   Here the infostruct is filled with information about the
 *   kernel.
 *   @param infostruct  a pointer to a structure filled with zeros
 */
void bi_getinfo(bi_info * infostruct)
{
   int i = 0, j = 0; /* loop var for functionCount */
   /* get environment variables for the kernel */
   evaluate_environment();
   infostruct->codesequence = bi_strdup("work_[1|2]()");
   infostruct->xaxistext = bi_strdup("Problem Size");
   infostruct->num_measurements = (MAX-MIN+1)/INCREMENT;
   if((MAX-MIN+1) % INCREMENT != 0)
     infostruct->num_measurements++;
   infostruct->num_processes = 1;
   infostruct->num_threads_per_process = 0;
//   infostruct->base_xaxis = 0;

   infostruct->kernel_execs_mpi1 = 0;
   infostruct->kernel_execs_mpi2 = 0;
   infostruct->kernel_execs_pvm = 0;
   infostruct->kernel_execs_omp = 0;
   infostruct->kernel_execs_pthreads = 0;
   /* B ########################################################*/
   functionCount = 2; /* number versions of this algorithm (ijk, ikj, kij, ... = 6 */
   valuesPerFunction = 2; /* time measurement and FLOPS (calculated) */
   /*########################################################*/
   infostruct->numfunctions = functionCount * valuesPerFunction;

   /* allocating memory for y axis texts and properties */
   allocYAxis(infostruct);
   /* setting up y axis texts and properties */
   for (j = 0; j < functionCount; j++)
   {
      /* B ########################################################*/
      int index1 = 0 * functionCount + j;
      int index2 = 1 * functionCount + j;
      //int index3 = 2 * functionCount + j;
      // 1st function
      infostruct->yaxistexts[index1] = bi_strdup("s");
      infostruct->selected_result[index1] = SELECT_RESULT_LOWEST;
      infostruct->base_yaxis[index1] = 0;
      // 2nd function
      infostruct->yaxistexts[index2] = bi_strdup("FLOPS");
      infostruct->selected_result[index2] = SELECT_RESULT_HIGHEST;
      infostruct->base_yaxis[index2] = 0;
      /*########################################################*/
      // 3rd function
      //infostruct->yaxistexts[index3] = bi_strdup("");
      //infostruct->selected_result[index3] = 0;
      //infostruct->base_yaxis[index3] = 0;
      switch (j)
      {
         /* B ########################################################*/
         case 1: // 2nd version legend text; maybe (ikj)
            infostruct->legendtexts[index1] =
               bi_strdup("Calculation Time (CPU)"); // "... (ikj)"
            infostruct->legendtexts[index2] =
               bi_strdup("FLOPS (CPU)"); // "... (ikj)"
            break;
         case 0: // 1st version legend text; maybe (ijk)
         default:
            infostruct->legendtexts[index1] =
               bi_strdup("Calculation Time (GPU)"); // "... (ijk)"
            infostruct->legendtexts[index2] =
               bi_strdup("FLOPS (GPU)"); // "... (ijk)"
         /*########################################################*/
      }
   }
   if (DEBUGLEVEL > 3)
   {
      /* the next for loop: */
      /* this is for your information only and can be ereased if the kernel works fine */
      for (i = 0; i < infostruct->numfunctions; i++)
      {
         printf("yaxis[%2d]=%s\t\t\tlegend[%2d]=%s\n",
            i, infostruct->yaxistexts[i], i, infostruct->legendtexts[i]);
      }
   }
}

/** Implementation of the bi_init of the BenchIT interface.
 *  Here you have the chance to allocate the memory you need.
 *  It is also possible to allocate the memory at the beginning
 *  of every single measurement and to free the memory thereafter.
 *  But always making use of the same memory is faster.
 *  HAVE A LOOK INTO THE HOWTO !
 */
void* bi_init(int problemSizemax)
{
   problemSizemax = MIN + (problemSizemax - 1) * INCREMENT;
   mydata_t* mdp;
   mdp = (mydata_t*)malloc(sizeof(mydata_t));
   if (mdp == 0)
   {
      fprintf(stderr, "Allocation of structure mydata_t failed\n"); fflush(stderr);
      exit(127);
   }
   size_t arraySize=problemSizemax * sizeof(int);
   mdp->a_host = (int*) malloc(arraySize);
   mdp->b_host = (int*) malloc(arraySize);
   CUDA_CHECK(hipMalloc((void**)&(mdp->a_device),arraySize));
   CUDA_CHECK(hipMalloc((void**)&(mdp->b_device),arraySize));
   int i;
   for (i=0;i<problemSizemax;i++) mdp->a_host[i]=i;
   CUDA_CHECK(hipMemcpy(mdp->a_device,mdp->a_host,arraySize,hipMemcpyHostToDevice));
   /*########################################################*/
   /* malloc our own arrays in here */
   /*########################################################*/
   return (void*)mdp;
}

/** The central function within each kernel. This function
 *  is called for each measurement step seperately.
 *  @param  mdpv         a pointer to the structure created in bi_init,
 *                       it is the pointer the bi_init returns
 *  @param  problemSize  the actual problemSize
 *  @param  results      a pointer to a field of doubles, the
 *                       size of the field depends on the number
 *                       of functions, there are #functions+1
 *                       doubles
 *  @return 0 if the measurement was sucessfull, something
 *          else in the case of an error
 */
int bi_entry(void* mdpv, int problemSize, double* results)
{
  /* timeInSecs: the time for a single measurement in seconds */
  double timeInSecs = 0.0;
  /* flops stores the calculated FLOPS */
  double flops = 0.0;
  /* j is used for loop iterations */
  int j = 0;
  /* cast void* pointer */
  mydata_t* mdp = (mydata_t*)mdpv;

  /* calculate real problemSize */
  problemSize = MIN + (problemSize - 1) * INCREMENT;

  /* check wether the pointer to store the results in is valid or not */
  if (results == NULL) return 1;

  /*########################################################*/
  /* maybe some init stuff in here */
  /*########################################################*/
  
  // the xaxis value needs to be stored only once!
  results[0] = (double)problemSize;

  for (j = 0; j < functionCount; j++)
  {
    /* B ########################################################*/
    int index1 = 0 * functionCount + j;
    int index2 = 1 * functionCount + j;
    /* choose version of algorithm */
    switch (j) {
      case 1: // 2nd version legend text; maybe (ikj)
        /* take start time, do measurement, and take end time */
        bi_startTimer(); moveCPU(mdp->a_host, mdp->b_host, problemSize); timeInSecs=bi_stopTimer();
        break;
      case 0: // 1st version legend text; maybe (ijk)
      default:
        CUDA_CHECK(hipMemcpy(mdp->a_device,mdp->a_host,problemSize*sizeof(int),hipMemcpyHostToDevice));
        int blocksize=min(problemSize,256);
  			dim3 dimBlock(blocksize);
  			dim3 dimGrid(ceil(problemSize/(float)blocksize));
        /* take start time, do measurement, and take end time */
        bi_startTimer();
        CUDA_CHECK_KERNEL_SYNC(moveGPU<<<dimGrid,dimBlock>>>(mdp->a_device,mdp->b_device,problemSize));
        timeInSecs=bi_stopTimer();
        CUDA_CHECK(hipMemcpy(mdp->b_host,mdp->b_device,problemSize*sizeof(int),hipMemcpyDeviceToHost));
        //maybe check results here
    }
    /* calculate the used time and FLOPS */
    /* If the operation was too fast to be measured by the timer function,
     * mark the result as invalid */
    if(timeInSecs == INVALID_MEASUREMENT){
    	flops = INVALID_MEASUREMENT;
    }else{
      // this flops value is a made up! this calulations should be replaced
      // by something right for the choosen algorithm
      flops = (double)problemSize;
    }
    /* store the results in results[1], results[2], ...
    * [1] for the first function, [2] for the second function
    * and so on ...
    * the index 0 always keeps the value for the x axis
    */
    /* B ########################################################*/
    results[index1 + 1] = timeInSecs;
    results[index2 + 1] = flops;
    /*########################################################*/
  }

  return 0;
}

/** Clean up the memory
 */
void bi_cleanup(void* mdpv)
{
   mydata_t* mdp = (mydata_t*)mdpv;
   /*########################################################*/
   /* may be freeing our arrays here */
   /*########################################################*/
   free(mdp->a_host);
   free(mdp->b_host);
   CUDA_CHECK(hipFree(mdp->a_device));
   CUDA_CHECK(hipFree(mdp->b_device));   
   if (mdp) free(mdp);
   return;
}
/********************************************************************/
/*************** End of interface implementations *******************/
/********************************************************************/

/* Reads the environment variables used by this kernel. */
void evaluate_environment()
{
   int errors = 0;
   char * p = 0;
   p = bi_getenv("BENCHIT_KERNEL_PROBLEMSIZE_MIN", 0);
   if (p == 0) errors++;
   else MIN = atoi(p);
   p = bi_getenv("BENCHIT_KERNEL_PROBLEMSIZE_MAX", 0);
   if (p == 0) errors++;
   else MAX = atoi(p);
   p = bi_getenv("BENCHIT_KERNEL_PROBLEMSIZE_INCREMENT", 0);
   if (p == 0) errors++;
   else INCREMENT = atoi(p);
   if (errors > 0)
   {
      fprintf(stderr, "There's at least one environment variable not set!\n");
      fprintf(stderr, "This kernel needs the following environment variables:\n");
      fprintf(stderr, "BENCHIT_KERNEL_PROBLEMSIZE_MIN\n");
      fprintf(stderr, "BENCHIT_KERNEL_PROBLEMSIZE_MAX\n");
      fprintf(stderr, "BENCHIT_KERNEL_PROBLEMSIZE_INCREMENT\n");
      fprintf(stderr, "\nThis kernel will iterate from BENCHIT_KERNEL_PROBLEMSIZE_MIN\n\
to BENCHIT_KERNEL_PROBLEMSIZE_MAX, incrementing by\n\
BENCHIT_KERNEL_PROBLEMSIZE_INCREMENT with each step.\n");
      exit(1);
   }
}
